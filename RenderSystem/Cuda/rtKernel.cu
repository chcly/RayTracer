#include "hip/hip_runtime.h"
/*
-------------------------------------------------------------------------------
    Copyright (c) Charles Carley.

  This software is provided 'as-is', without any express or implied
  warranty. In no event will the authors be held liable for any damages
  arising from the use of this software.

  Permission is granted to anyone to use this software for any purpose,
  including commercial applications, and to alter it and redistribute it
  freely, subject to the following restrictions:

  1. The origin of this software must not be misrepresented; you must not
     claim that you wrote the original software. If you use this software
     in a product, an acknowledgment in the product documentation would be
     appreciated but is not required.
  2. Altered source versions must be plainly marked as such, and must not be
     misrepresented as being the original software.
  3. This notice may not be removed or altered from any source distribution.
-------------------------------------------------------------------------------
*/
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <cstdint>
#include <cstdio>
#include "RenderSystem/Cuda/rtCuMath.inl"
#include "RenderSystem/Cuda/rtCudaUtils.h"
#include "RenderSystem/Data/rtCameraTypes.h"
#include "RenderSystem/Data/rtLightTypes.h"
#include "RenderSystem/Data/rtMaterialTypes.h"
#include "RenderSystem/Math/rtColor.h"
#include "RenderSystem/rtRenderSystem.h"

RT_DEVICE_API bool rtCudaRayIntersectsObject(rtObjectType* obj, rtCudaRay* ray, const rtVector2& lim)
{
    switch (obj->type)
    {
    case RT_AO_SHAPE_CUBE:
    case RT_AO_BVO:
        return rtCuBoxTest(obj->bounds.bMin, obj->bounds.bMax, *ray, lim);
    case RT_AO_SHAPE_SPHERE:
    {
        rtSphereVolume* sphere = (rtSphereVolume*)obj->bounds.data;
        if (sphere)
            return rtCuSphereTest(sphere->center, sphere->radius, *ray, lim);
        return false;
    }
    default:
        return false;
    }
}

RT_DEVICE_API bool rtCudaRayIntersectsObject(rtCudaHitResult* nearest, rtObjectType* obj, rtCudaRay* ray, const rtVector2& lim)
{
    switch (obj->type)
    {
    case RT_AO_SHAPE_CUBE:
    case RT_AO_BVO:
        return rtCuBoxTest(nearest, obj->bounds.bMin, obj->bounds.bMax, *ray, lim);
    case RT_AO_SHAPE_SPHERE:
    {
        rtSphereVolume* sphere = (rtSphereVolume*)obj->bounds.data;
        if (sphere)
            return rtCuSphereTest(nearest, sphere->center, sphere->radius, *ray, lim);
        return false;
    }
    default:
        return false;
    }
}

RT_DEVICE_API bool rtCudaTestScene(const rtSceneType* sc, rtCudaHitResult* nearest, rtCudaRay* ray, bool first = false)
{
    //SK_ASSERT(sc && sc->camera);

    // copy the limits..
    rtVector2 lim = sc->camera->limits;

    nearest->object = nullptr;

    for (uint32_t i = 0; i < sc->objects.size; ++i)
    {
        rtObjectType* obj = (rtObjectType*)sc->objects.data[i];

        if (rtCudaRayIntersectsObject(nearest, obj, ray, lim))
        {
            if (first)
            {
                nearest->object = obj;
                return true;
            }

            if (nearest->distance < lim.y)
            {
                nearest->object = obj;
                lim.y           = nearest->distance;
            }
        }
    }
    return nearest->object != nullptr;
}

RT_DEVICE_API bool rtCudaTestScene(const rtSceneType* sc, rtCudaRay* ray, const rtVector2 lim)
{
    for (uint32_t i = 0; i < sc->objects.size; ++i)
    {
        rtObjectType* obj = (rtObjectType*)sc->objects.data[i];

        if (rtCudaRayIntersectsObject(obj, ray, lim))
            return true;
    }
    return false;
}

RT_DEVICE_API void rtCuGetRayDirection(rtCudaRay*       dest,
                                       const rtScalar&  x,
                                       const rtScalar&  y,
                                       const rtVector4& rotation,
                                       const rtVector4& offset)
{
    rtVector3 d;
    d.x = offset.w * (x + RT_EPSILON) * offset.y;
    d.y = offset.w * (y + RT_EPSILON) * offset.y;
    d.z = -1;
    rtCuMulQuat(dest->direction, d, rotation);
}

RT_DEVICE_API rtScalar rtCuGetIntensity(rtLightType* light, const rtVector3& point)
{
    const skScalar x2 = rtCuFloat3DistSqu(light->location, point);
    if (light->mode == 0)
    {
        // https://www.desmos.com/calculator/l2ipt7pc9k
        // this is meant to not have to modify x,
        // ie x = sqrt(x), ... then x*x,
        // because it needs to be done here per pixel...
        const skScalar x2R2 = 16 * x2 * light->elevation;
        return light->rr / (light->rre4 + x2R2);
    }
    return (light->elevation * light->elevation * light->energy) / (x2 + 10e-3f);

}

RT_DEVICE_API void rtCuTraceColorAndLight(rtVector3&             pixel,
                                          const rtSceneType*     sc,
                                          const rtCudaHitResult* nearest,
                                          const rtCudaRay&       ray,
                                          const rtScalar&        v)
{
    if (nearest && nearest->object)
    {
        rtMaterialType* ma = nearest->object->material;

        if (ma->flags != RT_MA_SHADELESS)
        {
            // apply lights
            if (sc->lights.size <= 0)
                return;

            rtVector3 world{};
            rtCuMix3(&world, &sc->horizon, &sc->zenith, 0.5f);

            rtScalar kd = 0;
            rtScalar nd = 0;
            rtScalar ks = 0;
            rtScalar sh = 1;
            rtScalar ia = 0;

            for (uint32_t l = 0; l < sc->lights.size; ++l)
            {
                rtLightType* li = sc->lights.data[l];

                rtVector3      lv = rtCuFloat3Norm(li->location, nearest->point);
                const rtScalar ln = rtCuFloat3Dot(lv, nearest->normal);

                const rtScalar i = rtCuGetIntensity(li, nearest->point);
                ia += ma->ambient;

                if (ln > 10e-4f)
                {
                    kd += ma->diffuse * ln * i;

                    if (ma->flags & RT_MA_SPECULAR && ma->specular > 0)
                    {
                        const rtVector3 eye = rtCuFloat3Norm(ray.origin, nearest->point);
                        const rtVector3 rfl = rtCuReflectFloat3(lv, nearest->normal);

                        const skScalar d = rtCuFloat3Dot(rfl, eye);
                        if (d > 10e-3f)
                            ks += ma->specular * i * powf(d, ma->hardness);
                    }

                    if (ma->flags & RT_MA_SHADOW)
                    {
                        rtCudaRay       r{nearest->point, lv};
                        const rtVector2 lim = {10e-3f, 1000};

                        if (rtCudaTestScene(sc, &r, lim))
                        {
                            const rtScalar fac = .5f * (1.f + i);
                            if (fac < 1.f)
                                sh *= fac;
                        }
                    }
                }
                else
                    nd += ma->diffuse * -ln * i;
            }

            ia /= rtScalar(sc->lights.size);

            world   = rtCuMulFloat3(world, ia);
            pixel.x = ma->color.x;
            pixel.y = ma->color.y;
            pixel.z = ma->color.z;
            pixel   = rtCuMulFloat3v(pixel, world);
            pixel   = rtCuMulFloat3(pixel, sh);
            pixel   = rtCuMulFloat3(pixel, kd);
            pixel   = rtCuAddFloat3(pixel, ks);

            rtCuMix(&pixel, &world, nd * 0.125f);

            rtCuSaturate(&pixel);
        }
        else
        {
            pixel.x = ma->color.x;
            pixel.y = ma->color.y;
            pixel.z = ma->color.z;
        }
    }
    else
    {
        rtScalar t = 1 - v;
        rtCuSaturate(&t);
        const rtScalar it = 1 - t;

        pixel.x = sc->horizon.x * t + sc->zenith.x * it;
        pixel.y = sc->horizon.y * t + sc->zenith.y * it;
        pixel.z = sc->horizon.z * t + sc->zenith.z * it;
    }
}

RT_DEVICE_API void rtCuSetPixel(rtCudaTarget*    target,
                                const int32_t    x,
                                const int32_t    y,
                                const rtVector3& bg)
{
    const int32_t loc = x * 4 + y * target->pitch;
    if (loc > 0 && loc < target->length)
    {
        uint8_t* pix = &target->pixels[loc];

        pix[0] = (uint8_t)(bg.z * 255.0f);
        pix[1] = (uint8_t)(bg.y * 255.0f);
        pix[2] = (uint8_t)(bg.x * 255.0f);
        pix[3] = 255;
    }
}

RT_DEVICE_API void rtCpuRenderSample(rtVector3&          pixel,
                                     rtCudaHitResult&    nearest,
                                     const rtCameraType* ca,
                                     const rtSceneType*  sc,
                                     const rtScalar&     kX,
                                     const rtScalar      kY,
                                     const int           y)
{
    rtCudaRay ray{};
    ray.origin = ca->location;
    rtCuGetRayDirection(&ray, kX, kY, ca->rotation, ca->offset);

    if (rtCudaTestScene(sc, &nearest, &ray))
    {
        // process hits
        if (sc->flags & RM_COLOR_AND_LIGHT)
            rtCuTraceColorAndLight(pixel, sc, &nearest, ray, 0);

        if (sc->flags & RM_COMPUTED_NORMAL)
        {
            if (sc->flags == RM_COMPUTED_NORMAL)
            {
                pixel = {
                    0.5f + nearest.normal.x * 0.5f,
                    0.5f + nearest.normal.y * 0.5f,
                    0.5f + nearest.normal.z * 0.5f,
                };
            }
            else
            {
                pixel.x = 0.125f + pixel.x * 0.5f + nearest.normal.x * 0.25f;
                pixel.y = 0.125f + pixel.y * 0.5f + nearest.normal.y * 0.25f;
                pixel.z = 0.125f + pixel.z * 0.5f + nearest.normal.z * 0.25f;
            }
        }

        if (sc->flags & RM_DISTANCE)
        {
            const skScalar df = skScalar(2.f) / (nearest.distance + skScalar(10e-3));
            if (sc->flags == RM_DISTANCE)
                pixel = {df, df, df};
            else
            {
                pixel.x = (pixel.x + df) * 0.5f;
                pixel.y = (pixel.y + df) * 0.5f;
                pixel.z = (pixel.z + df) * 0.5f;
            }
        }
    }
    else
    {
        // process misses
        if (sc->flags & RM_COLOR_AND_LIGHT)
            rtCuTraceColorAndLight(pixel, sc, &nearest, ray, rtScalar(y) * ca->offset.y);
    }

    // limit and invert...
    rtCuSaturate(&pixel);

    if (sc->flags & RM_INVERT)
    {
        pixel.x = 1.f - pixel.x;
        pixel.y = 1.f - pixel.y;
        pixel.z = 1.f - pixel.z;
    }
}

RT_KERNEL_API void rtCuRenderKernel(rtSceneType*  sc,
                                    rtCudaTarget* target)
{
    const int32_t start  = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    const rtCameraType* ca = sc->camera;
    rtCudaHitResult     nearest{};

    for (int32_t i = start; i < target->dimensions; i += stride)
    {
        const int32_t x = i / target->height;
        const int32_t y = i % target->height;

        if (!(sc->flags & RM_AA))
        {
            const skScalar kX = skScalar(target->width - 2 * x);
            const skScalar kY = skScalar(target->height - 2 * y);

            rtVector3 curPixel{};

            rtCpuRenderSample(curPixel, nearest, ca, sc, kX, kY, y);
            rtCuSetPixel(target, x, target->height - 1 - y, curPixel);
        }
        else
        {
            const skScalar kX = skScalar(target->width - 2 * x);
            const skScalar kY = skScalar(target->height - 2 * y);

            rtVector3      P[5]{};
            const skScalar h = 1.1625f;

            rtCpuRenderSample(P[0], nearest, ca, sc, kX, kY, y);
            rtCpuRenderSample(P[1], nearest, ca, sc, kX - h, kY - h, y);
            rtCpuRenderSample(P[2], nearest, ca, sc, kX + h, kY + h, y);
            rtCpuRenderSample(P[3], nearest, ca, sc, kX + h, kY - h, y);
            rtCpuRenderSample(P[4], nearest, ca, sc, kX - h, kY + h, y);

            rtVector3 curPixel{};
            curPixel.x = (P[0].x + P[1].x + P[2].x + P[3].x + P[4].x) * 0.2f;
            curPixel.y = (P[0].y + P[1].y + P[2].y + P[3].y + P[4].y) * 0.2f;
            curPixel.z = (P[0].z + P[1].z + P[2].z + P[3].z + P[4].z) * 0.2f;

            rtCuSetPixel(target, x, target->height - 1 - y, curPixel);
        }
    }
}

RT_CU_API void rtCudaKernelMain(rtCudaFrameBuffer* fb, rtCudaTarget* buffer, rtSceneType* scene)
{
    hipError_t status;
    rtCuRenderKernel<<<fb->threads, fb->blocks>>>(scene, buffer);

    if ((status = hipDeviceSynchronize()) != hipSuccess)
        printf("Sync failed %d\n", status);
}
